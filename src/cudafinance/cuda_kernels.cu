#include <hip/hip_runtime.h>

#include <stdio.h>

#define CUDA_CHECK(call) \
    do { \
        if((call) != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d\n", __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void computeSMA(const float *input, float *output, int numElements, int windowSize) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        float sum = 0.0;
        int count = 0;
        for (int j = max(0, i - windowSize + 1); j <= i; ++j) {
            sum += input[j];
            count++;
        }
        output[i] = sum / count;
    }
}

void launchSMA_CUDA(const float* h_input, float* h_output, int numElements, int windowSize) {
    float *d_input = NULL, *d_output = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_input, numElements * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&d_output, numElements * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, numElements * sizeof(float), hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    computeSMA<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, numElements, windowSize);

    CUDA_CHECK(hipMemcpy(h_output, d_output, numElements * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
}
